#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY are used to set the number of threads in a CUDA block 
#define TILEX 32
#define TILEY 16
#if TILEX > TILEY
	#define TILEZ	TILEX
#else
	#define TILEZ	TILEY
#endif

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {
	__shared__ float Mds[TILEY][TILEZ];
	__shared__ float Nds[TILEZ][TILEX];
	
	int r = by * TILEY + ty;
	int c = bx * TILEX + tx;
	
	float tmp = 0;
	for(int m=0; m < n/TILEZ; m++){
		for(int i=0; i < TILEZ/TILEX; i++)
			//Mds[ty][m*TILEZ + tx*(TILEZ/TILEX) + i] = ad[r * n + (m*TILEZ + tx*(TILEZ/TILEX) + i)]; 
			Mds[ty][tx*(TILEZ/TILEX) + i] = ad[r * n + (m*TILEZ + tx*(TILEZ/TILEX) + i)]; 
		for(int i=0; i < TILEZ/TILEY; i++)
			//Nds[m*TILEZ + ty*(TILEZ/TILEY) + i][tx] = bd[(m*TILEZ + ty*(TILEZ/TILEY) + i)*n + c];
			Nds[ty*(TILEZ/TILEY) + i][tx] = bd[(m*TILEZ + ty*(TILEZ/TILEY) + i)*n + c];
		__syncthreads();
		for(int k=0; k<TILEZ; k++){
			tmp += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads();
	}
	cd[r*n+c]=tmp;
}
