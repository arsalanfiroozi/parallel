#include "hip/hip_runtime.h"
// ONLY MODIFY THIS FILE

#include "scan2.h"
#include "gpuerrors.h"
#include "hip/hip_runtime.h"
#include ""

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

__global__ void kernelFunc(float *ad, float *cd, float *ed, const int n)
{
    // This kernel implements scan algorithm in a way that in the end we have scan of each block (1024 cells) 
    __shared__ float Mds[1024];

    int r = bx * 1024 + tx;
    Mds[tx] = ad[r]; // Copy to shared Memory
    // ad[r] = tx + 1;  // For debug purposes.
    __syncthreads();

    // In this part the first stage of the tree in Blelloch algorithm is calculated.
    int j = 2;
    for (; j <= n; j = j * 2)
    {
        if ((r + 1) % j == 0)
            Mds[tx] = Mds[tx] + Mds[tx - j / 2];
        __syncthreads();
    }

    // Replace the last element in each block by 0 and also store it for later. This is done in the last thread of 
    // each block and is dependent on the number of threads.
    float e;
    if (n >= 1024 && tx == 1023)
    {
        e = Mds[tx];
        Mds[tx] = 0;
    }
    else if (n < 1024 && tx == n - 1)
    {
        e = Mds[tx];
        Mds[tx] = 0;
    }
    __syncthreads();

    // In this part the second stage of the tree in Blelloch algorithm is calculated.
    j = n;
    for (; j >= 2; j = j / 2)
    {
        if ((r + 1) % j == 0)
        {
            float t = Mds[tx];
            Mds[tx] = Mds[tx] + Mds[tx - j / 2];
            Mds[tx - j / 2] = t;
        }
        __syncthreads();
    }
    
    // For converting exclusive scan to inclusive scan, we need to have a shift to left and then the last element
    // should be replaced by total sum.
    if (tx >= 1)
        cd[r - 1] = Mds[tx];
    if (n >= 1024 && tx == 1023)
        cd[r] = e;
    else if (n < 1024 && tx == n - 1)
        cd[r] = e;
    __syncthreads();
    
    // The last element of each block shoud be stored as a output in order to add them to the next blocks. 
    // So that the scan algorithm is completed.
    if (tx == 1023)
    {
        ed[bx] = cd[r];
    }
}

__global__ void kernelFunc2(float *ed, float *cd, const int n)
{
    // This kernel is used to add sum of all elements in previous blocks in order to complete scan algrorithm and
    // compensate the effect of having seperate blocks in kernelFunc
    int r = bx * 1024 + tx;

    if (bx > 0)
    {
        cd[r] += 1 * ed[bx - 1];
    }
}

dim3 getDimGrid(const int n)
{
    // We need enough blocks to cover all cells of input vector so if all of them are fit into one block, we need one and if not
    // we need n/1024 due to the maximum possible number of threads in each block.
    if (n < 1024)
    {
        dim3 dimGrid(1, 1);
        return dimGrid;
    }
    else
    {
        dim3 dimGrid(n / 1024, 1);
        return dimGrid;
    }
}
dim3 getDimBlock(const int n)
{
    // If all of them are fit into one block, we need one block with enough threads and if not 
    // 1024 threads are specified due to the maximum possible number of threads in each block.
    if (n < 1024)
    {
        dim3 dimBlock(n, 1);
        return dimBlock;
    }
    else
    {
        dim3 dimBlock(1024, 1);
        return dimBlock;
    }
}

void gpuKernel(float *a, float *c, int n)
{
    // This function is run in the CPU. This is a recursive function that has 2 section:
    // 1. Compute scan of input vector 
    // 2. Compute scan of the last element of each block by the "gpuKernel" recursively
    // 3. Add computed scan of the last elements to blocks.
    // For n >= 2^26 the output is calculated by dividing into two subvector with size n/2 by
    // calculating scan of each of them serially. in order to link the first subvector to second subvector,
    // I added the last element of the calculated scan of first subvector to the first element of the second input subvector.
    // So for all m={1, ..., 20, 21, ..., 29} the output is correct.
    if (n < (1 << 26))
    {
        float *ad;
        float *cd;
        float *ed;
        float *e = (float *)malloc(n / 1024 * sizeof(float));

        // Phase 1
        HANDLE_ERROR(hipMalloc((void **)&ad, n * sizeof(float)));
        HANDLE_ERROR(hipMalloc((void **)&cd, n * sizeof(float)));
        HANDLE_ERROR(hipMalloc((void **)&ed, (n / 1024) * sizeof(float)));

        HANDLE_ERROR(hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice));

        dim3 dimGrid = getDimGrid(n);  
        dim3 dimBlock = getDimBlock(n);

        if (n < 1024)
            kernelFunc<<<dimGrid, dimBlock>>>(ad, cd, ed, n);
        else
            kernelFunc<<<dimGrid, dimBlock>>>(ad, cd, ed, 1024);

        HANDLE_ERROR(hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(e, ed, n / 1024 * sizeof(float), hipMemcpyDeviceToHost));

        HANDLE_ERROR(hipFree(ad));
        HANDLE_ERROR(hipFree(cd));
        HANDLE_ERROR(hipFree(ed));

        float *e_s;
        if (n > 1024)
        {
            //printf("Hit!\n");
            e_s = (float *)malloc(n / 1024 * sizeof(float));
            gpuKernel(e, e_s, n / 1024);
        }
        else
        {
            e_s = e;
        }

        // Phase 2
        HANDLE_ERROR(hipMalloc((void **)&ed, n / 1024 * sizeof(float)));
        HANDLE_ERROR(hipMalloc((void **)&cd, n * sizeof(float)));

        HANDLE_ERROR(hipMemcpy(ed, e_s, n / 1024 * sizeof(float), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(cd, c, n * sizeof(float), hipMemcpyHostToDevice));

        dimGrid = getDimGrid(n);
        dimBlock = getDimBlock(n);

        kernelFunc2<<<dimGrid, dimBlock>>>(ed, cd, n);

        HANDLE_ERROR(hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost));

        HANDLE_ERROR(hipFree(cd));
        HANDLE_ERROR(hipFree(ed));
    }
    else
    {
        gpuKernel(a, c, n / 2);
        a[n / 2] += c[n / 2 - 1];
        gpuKernel(a + n / 2, c + n / 2, n / 2);
    }
}
