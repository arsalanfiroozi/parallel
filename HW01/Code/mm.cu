#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE

#include "mm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may change the following values
#define TILEX 16
#define TILEY 16

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}

//-----------------------------------------------------------------------------
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m,const int n) {
	// write your GPU kernel function here
	// note that maximum # of threads per block is 1024
	int i = bx*blockDim.x+tx; // TILEX-1 * n/TILEX + n/TILEX - 1 = n - 1
	int j = by*blockDim.y+ty;
	int k;
	cd[n*j+i]=0;
	for(k=0;k<n;k++){
		cd[n*j+i] += ad[n*j+k]*bd[n*k+i];
	}
}
